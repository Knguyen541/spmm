// https://subscription.packtpub.com/book/programming/9781788996242/1/ch01lvl1sec03/hello-world-from-cuda


#include <hip/hip_runtime.h>
#include<stdio.h>
#include<stdlib.h>

__global__ void print_from_gpu(void) {
    printf("Hello World! from thread [%d,%d] \
        From device\n", threadIdx.x,blockIdx.x);
}

int main(void) {
    printf("Hello World! from host!\n");
    print_from_gpu<<<1,1>>>();
    hipDeviceSynchronize();
    return 0;
}